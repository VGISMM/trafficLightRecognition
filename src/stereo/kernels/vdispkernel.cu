#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "vdispkernel.h"

__global__
void vdispKernel(unsigned char* const image,
                       unsigned int* vdispImage,
                       int numRows, int numCols, int vdispNumRows, int vdispNumCols)
{
  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;

  // map the two 2D indices to a single linear, 1D index
  int grid_width = gridDim.x * blockDim.x;
  int index = index_y * grid_width + index_x;
  
  //vdispImage[index] = image[index];
  
  int vdispIdx = (index/(numCols-1)*255) + image[index];
  atomicAdd(&vdispImage[vdispIdx],1);

  /*
  if(image[index]<240)
  {
    int vdispIdx = (index/(numCols-1)*255) + image[index];
    // write out the final result
    if (vdispImage[vdispIdx] < 255)
    {
      atomicAdd(&vdispImage[vdispIdx],1);
    }
  }
  */

  
}

void launchVdispKernel(unsigned char* const d_image,
                            unsigned int* d_vdispImage, 
                            size_t numRows, size_t numCols, size_t vdispNumRows, size_t vdispNumCols)
{
  const int thread = 32;
  const dim3 blockSize( thread, thread, 1);
  const dim3 gridSize( ceil(numRows/(float)thread), ceil(numCols/(float)thread), 1);
  
  //rgba_to_greyscale<<<((numRows*numCols)/32), 32>>>(d_image, d_vdispImage, numRows, numCols, vdispNumRows, vdispNumCols);
  vdispKernel<<<gridSize, blockSize>>>(d_image, d_vdispImage, numRows, numCols, vdispNumRows, vdispNumCols);
  hipDeviceSynchronize();
}