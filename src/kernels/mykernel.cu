#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <algorithm>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;
__global__
void rgba_to_greyscale(unsigned char* const image,
                       unsigned int* vdispImage,
                       int numRows, int numCols, int vdispNumRows, int vdispNumCols)
{
  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;

  // map the two 2D indices to a single linear, 1D index
  int grid_width = gridDim.x * blockDim.x;
  int index = index_y * grid_width + index_x;
  
  int vdispIdx = (index/(numCols-1)*255) + image[index];
  // write out the final result
  //if (vdispImage[vdispIdx] < 255)
  //{
    atomicAdd(&vdispImage[vdispIdx],1);
    //vdispImage[vdispIdx] = vdispImage[vdispIdx]+1;
  //}
  
}

void your_rgba_to_greyscale(unsigned char* const d_image,
                            unsigned int* d_vdispImage, 
                            size_t numRows, size_t numCols, size_t vdispNumRows, size_t vdispNumCols)
{
  const int thread = 32;
  const dim3 blockSize( thread, thread, 1);
  const dim3 gridSize( ceil(numRows/(float)thread), ceil(numCols/(float)thread), 1);
  
  //rgba_to_greyscale<<<((numRows*numCols)/32), 32>>>(d_image, d_vdispImage, numRows, numCols, vdispNumRows, vdispNumCols);
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_image, d_vdispImage, numRows, numCols, vdispNumRows, vdispNumCols);
  hipDeviceSynchronize();
}
